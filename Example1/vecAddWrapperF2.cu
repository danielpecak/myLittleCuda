#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

extern "C" void vecadd_wrapper(int n, float a[n], float b[n], float c[n])
{
    // Device input vectors
    float *d_a;
    float *d_b;
    //Device output vector
    float *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
    // Number of threads in each thread block
    blockSize = 1024;
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
