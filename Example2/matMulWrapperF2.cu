
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Matrix multiplication: AxB=C

//CUDA kernel. Each thread takes care of one cell of C matrix
__global__ void matmul(double *a, double *b, double *c, int n)
{
  // Get global thread ID
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  int Row = blockIdx.y*blockDim.y+threadIdx.y;

  // Not out of bounds
  if((Col<n) && (Row<n)) {// Mutliply matrices
    // c[Row*n + Col] = 0;
    double sum = 0.0;
    for(int k=0;k<n;k++) {
      // c[Row*n + Col] += a[Row*n+k]*b[k*n+Col];
      sum += a[Row*n+k]*b[k*n+Col];
    }
    c[Row*n + Col] = sum;
  }
}

extern "C" void matmul_wrapper(int n, double h_a[], double h_b[], double h_c[])
{
  // Device input matrices
  double *d_a;
  double *d_b;
  // Device output matrices
  double *d_c;

  //Size, in bytes, of each array
  size_t bytes = n*n*sizeof(double);

  // Allocate memory for each matrix on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  printf(" C Memory allocated \n");

  // Copy host matrices to device
  hipMemcpy(d_a,h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b, bytes, hipMemcpyHostToDevice);
  printf(" C Data sent to GPU \n");

  int blockSize, gridSize;
  // Number of threads in each thread block
  blockSize = 32;
  // Number of thread blocks in grid
  gridSize = (int)ceil((double)n/blockSize);

  dim3 dimBlock(blockSize,blockSize);
  dim3 dimGrid(gridSize,gridSize);
  printf("   GridSize: %d\n", gridSize);
  printf("   BlockSize: %d\n", blockSize);

  // Execute the kernel
  matmul<<<dimGrid, dimBlock>>>(d_a,d_b,d_c, n);
  printf(" C Kernel executed \n");

  // Copy array back to host
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // CHECK RESULTS for 3x3 MATRIX
  // printf("%f %f %f\n",h_a[0],h_a[1],h_a[2]);
  // printf("%f %f %f\n",h_a[3],h_a[4],h_a[5]);
  // printf("%f %f %f\n",h_a[6],h_a[7],h_a[8]);
  // printf("\n");
  // printf("%f %f %f\n",h_b[0],h_b[1],h_b[2]);
  // printf("%f %f %f\n",h_b[3],h_b[4],h_b[5]);
  // printf("%f %f %f\n",h_b[6],h_b[7],h_b[8]);
  // printf("\n");
  // printf("%f %f %f\n",h_c[0],h_c[1],h_c[2]);
  // printf("%f %f %f\n",h_c[3],h_c[4],h_c[5]);
  // printf("%f %f %f\n",h_c[6],h_c[7],h_c[8]);

  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf(" C =============== \n");
}
